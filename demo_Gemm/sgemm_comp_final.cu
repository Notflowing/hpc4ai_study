#include "hip/hip_runtime.h"
// optimize sgemm

#include <stdio.h>
#include <cmath>
#include <stdlib.h>
#include "assert.h" 

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mma.h>

// cutlass
#include "cutlass/gemm/device/gemm.h"

using namespace nvcuda;

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				                                            \
{									                                                    \
    hipError_t e = (func);			                                                    \
    if(e != hipSuccess)						                                        \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}


__global__ void sgemm_naive(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
                           const int M, const int N, const int K) {
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    int ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < N && ty < M) {
        float sum = 0.0f;
        #pragma unroll
        for (int i = 0; i < K; ++i) {
            sum += A[ty * K + i] * B[i * N + tx];
        }
        C[ty * N + tx] = sum;
    }
}


#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 8

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void Sgemm( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];  //每个线程需要开启额外的寄存器进行存储
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    // 定位到该block处理的矩阵A,B的起始地址
    A = &A[(BLOCK_SIZE_M * by)* K];
    B = &B[BLOCK_SIZE_N * bx];

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();
    // load A from shared memory to register
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
        // 该线程需要处理的数据
        FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[0][0][THREAD_SIZE_Y * ty + thread_y]);
    }
    // load B from shared memory to register
    #pragma unroll
    for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
        FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[0][0][THREAD_SIZE_X * tx + thread_x]);
    }

    int write_stage_idx = 1;
    int tile_idx = 0;   // 大迭代时在A矩阵的列号
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K-1; ++j){
            // load next tile from shared mem to register 
            // load A from shared memory to register
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
                FETCH_FLOAT4(frag_a[(j+1)%2][thread_y]) = FETCH_FLOAT4(As[load_stage_idx][j+1][THREAD_SIZE_Y * ty + thread_y]);
            }
            // load B from shared memory to register
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
                FETCH_FLOAT4(frag_b[(j+1)%2][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx][j+1][THREAD_SIZE_X * tx + thread_x]);
            }
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; thread_y += 4) {
            FETCH_FLOAT4(frag_a[0][thread_y]) = FETCH_FLOAT4(As[load_stage_idx^1][0][THREAD_SIZE_Y * ty + thread_y]);
        }
        // load B from shared memory to register
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x += 4) {
            FETCH_FLOAT4(frag_b[0][thread_x]) = FETCH_FLOAT4(Bs[load_stage_idx^1][0][THREAD_SIZE_X * tx + thread_x]);
        }
        //compute last tile mma THREAD_SIZE_X x THREAD_SIZE_Y
        #pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
            #pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    }while(tile_idx< K);

    // store back to C
    #pragma unroll
    for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
        #pragma unroll
        for (int thread_x = 0; thread_x < THREAD_SIZE_X; thread_x+=4) {
            FETCH_FLOAT4(C[OFFSET(
                BLOCK_SIZE_M * by + ty * THREAD_SIZE_Y + thread_y,
                BLOCK_SIZE_N * bx + tx * THREAD_SIZE_X + thread_x,
                N)]) = FETCH_FLOAT4(accum[thread_y][thread_x]);
        }
    }
}

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
                                 int m_ld, int n_ld, int k_ld) {
    // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = n_ld;
    int ldc = n_ld;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major>
        a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major>
        b_frag;
    // wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(c_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < k_ld; i += WMMA_K) {
        int aCol = i;
        int aRow = warpM * WMMA_M;
        int bCol = warpN * WMMA_N;
        int bRow = i;

        // Bounds checking
        if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bCol + bRow * ldb, ldb);

#pragma unroll
            for (int t = 0; t < a_frag.num_elements; t++) {
                a_frag.x[t] = wmma::__float_to_tf32(a_frag.x[t]);
            }

#pragma unroll
            for (int t = 0; t < b_frag.num_elements; t++) {
                b_frag.x[t] = wmma::__float_to_tf32(b_frag.x[t]);
            }
            // Perform the matrix multiplication
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result
    // scaled by alpha
    int cCol = warpN * WMMA_N;
    int cRow = warpM * WMMA_M;

    if (cRow < m_ld && cCol < n_ld) {
        // Store the output
        wmma::store_matrix_sync(c + cCol + cRow * ldc, c_frag, ldc,
                            wmma::mem_row_major);
    }
    // printf("%f\t", c[cCol + cRow * ldc]);
}




int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    int M = atoi(argv[1]);
    int K = atoi(argv[2]);
    int N = atoi(argv[3]);

    assert( M%8 == 0); 
    assert( N%8 == 0); 
    assert( K%8 == 0); 

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);
    float* h_C2 = (float*)malloc(bytes_C);
    float* h_C3 = (float*)malloc(bytes_C);
    float* h_C4 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));
    double msecPerMatrixMul[5] = {0, 0, 0, 0, 0};
    double gigaFlops[5] = {0, 0, 0, 0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;
    const bool ENABLE_DOUBLE_BUFFER = true;

    // generate A
    for( int i = 0; i < M * K; i++ ){
        h_A[i] = i / 13;
        // h_A[i] = 1;
    }

    // generate B
    for( int i = 0; i < K * N; i++ ) {
        h_B[i] = i % 13;
        // h_B[i] = 1;
    }

    memset(h_C, 0, bytes_C);

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1000;

    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
    dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
    for (int run = 0 ; run < nIter; run ++ ) {
        Sgemm<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // tensor core mma
    dim3 gridDim;
    dim3 blockDim;

    // blockDim.x must be a multple of warpSize
    // 128x4 means we have 16 warps and a block computes a 64x64 output tile
    blockDim.x = 128;
    blockDim.y = 4;

    gridDim.x = (M + (WMMA_M * blockDim.x / 32 - 1)) /
                (WMMA_M * blockDim.x / 32);
    gridDim.y = (N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

    // printf("Computing... using simple_wmma_gemm kernel\n");
    // checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        simple_wmma_gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "Tsrcore Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    // cutlass
    int lda = K;
    int ldb = N;
    int ldc = N;
 
    float alpha = 1.0f;      //alpha
    float beta = 0.0f;       //beta

    using ColumnMajor = cutlass::layout::ColumnMajor;             // 列主序存储方式
    using RowMajor    = cutlass::layout::RowMajor;                // 行主序存储方式
     
    using CutlassGemm = cutlass::gemm::device::Gemm<float,        // A矩阵数据类型
                                                    RowMajor,     // A矩阵存储方式
                                                    float,        // B矩阵数据类型
                                                    RowMajor,     // B矩阵存储方式
                                                    float,        // C矩阵数据类型
                                                    RowMajor,     // C, D矩阵存储方式
                                                    float>;                      // Element type for internal accumulation
                                                    // cutlass::arch::OpClassSimt, // Operator class tag
                                                    // cutlass::arch::Sm80>;       // Tag indicating architecture to tune for
                                             
    CutlassGemm gemm_operator;                  // 声明cutlassgemm类
    CutlassGemm::Arguments args({M, N, K},      // Gemm Problem dimensions
                                {d_A, lda},     // source matrix A
                                {d_B, ldb},     // source matrix B
                                {d_C, ldc},     // source matrix C
                                {d_C, ldc},     // destination matrix D
                                {alpha, beta}); // alpha & beta

    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        gemm_operator(args); //运行Gemm 
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop)); 

    checkCudaErrors(hipMemcpy( h_C2, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[2] = msecTotal / nIter;
    gigaFlops[2] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[2] / 1000.0f);
    printf( "Cutlass Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
            gigaFlops[2],
            msecPerMatrixMul[2],
            flopsPerMatrixMul);

    // Naive Gemm
    dim3 blockNaive(16, 16, 1);
    dim3 gridNaive((N + blockNaive.x - 1)/ blockNaive.x, (M + blockNaive.y - 1) / blockNaive.y);
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        sgemm_naive<<<gridNaive, blockNaive>>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C3, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[3] = msecTotal / nIter;
    gigaFlops[3] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[3] / 1000.0f);
    printf( "NaiveGe Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[3],
        msecPerMatrixMul[3],
        flopsPerMatrixMul);

    // cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    // float alpha = 1.0;
    // float beta = 0;
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
            M, N, K, &alpha, 
            d_A, K, d_B, N, &beta, d_C, N
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // 得到的h_C4是正确结果的转置
    checkCudaErrors(hipMemcpy( h_C4, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[4] = msecTotal / nIter;
    gigaFlops[4] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[4] / 1000.0f);
    printf( "CuBlas  Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[4],
        msecPerMatrixMul[4],
        flopsPerMatrixMul);

    hipblasDestroy(blas_handle);
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = std::max({fabs(h_C[i] - h_C4[col * M + row]), fabs(h_C1[i] - h_C4[col * M + row]), 
                                   fabs(h_C2[i] - h_C4[col * M + row]), fabs(h_C3[i] - h_C4[col * M + row])});
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C[i], h_C4[col * M + row], eps);
            correct = false;
            break;
        }
        // if (abs_err > eps) {
        //     printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
        //             i, h_C[i], h_C1[col * M + row], eps);
        //     correct = false;
        //     break;
        // }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio_Optim= %f\n", gigaFlops[0] / gigaFlops[4]);
    printf("ratio_Tensr= %f\n", gigaFlops[1] / gigaFlops[4]);
    printf("ratio_Cutls= %f\n", gigaFlops[2] / gigaFlops[4]);
    printf("ratio_Naive= %f\n", gigaFlops[3] / gigaFlops[4]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
    free(h_C2);
    free(h_C3);
    free(h_C4);
}
